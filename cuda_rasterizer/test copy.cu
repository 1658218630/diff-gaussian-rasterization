#include "hip/hip_runtime.h"
/******************************************************************************
 * test.cu – runtime & gradient check for computeCov2DCUDA
 *
 * 编译示例（需要开启可分离编译才能跨 .cu 调用 device 函数）：
 *     nvcc -g -G -std=c++17 -arch=sm_70 test.cu forward.cu backward.cu \
 *          -o test -rdc=true
 *
 * 运行时创建 1 个高斯，对 computeCov2DCUDA 的解析梯度与
 * 前向数值差分做比较，给出 PASS / FAIL。
 *****************************************************************************/

 #define GLM_COMPILER 0
 #include <cstdio>
 #include <cmath>
 #include <hip/hip_runtime.h>
 #include <glm/glm.hpp>
 
 #include "forward.h"   // forward 侧的前向投影实现  :contentReference[oaicite:0]{index=0}
 #include "backward.h"  // backward 侧的 preprocess（里面会调用 computeCov2DCUDA）:contentReference[oaicite:1]{index=1}
 
 /*** ---------- forward 侧已有的 device 函数声明 --------- ***/
 struct MeanCov2D { float2 mean; float3 cov; };
 __device__ MeanCov2D computeMeanCov2D_statistical(
         const float3& mean_world,
         float focal_x, float focal_y,
         float tan_fovx, float tan_fovy,
         const float* cov3D,            // 6-float packed Σ₃ᴅ
         const float* viewmatrix);      // 4×4 row-major
 
 /* 把 forward 结果写出来，供有限差分用 */
 __global__ void ForwardKernel(
         int P,
         const float3* means,
         const float*  cov3Ds,
         float fx, float fy,
         float tan_fx, float tan_fy,
         const float* view,
         float4* out_conic)             // (μx, μy, σx², σy²)
 {
     int i = threadIdx.x + blockIdx.x * blockDim.x;
     if (i >= P) return;
 
     MeanCov2D mc = computeMeanCov2D_statistical(means[i], fx, fy,
                                                 tan_fx, tan_fy,
                                                 cov3Ds + 6*i, view);
     out_conic[i] = make_float4(mc.mean.x, mc.mean.y, mc.cov.x, mc.cov.z);
 }
 
 /*** --------- backward 侧核函数声明（在 backward.cu 定义） -------- ***/
 __global__ void computeCov2DCUDA(int P,
         const float3* means, const int* radii,
         const float*  cov3Ds,
         float fx, float fy,
         float tan_fx, float tan_fy,
         const float* view,
         const float* dL_dconics,
         float3* dL_dmeans,
         float*  dL_dcov);
 
 int main()
 {
     /* ---------- 1. 构造最小输入 ---------- */
     constexpr int P = 1;
     float3 h_means[P]   = { make_float3(0.f, 0.f, 5.f) };
     int    h_radii[P]   = { 1 };
     float  h_cov3Ds[6]  = { 0.1f, 0.f, 0.f, 0.1f, 0.f, 0.1f };   // 对称矩阵上三角
     float  h_view[16]   = { 1,0,0,0,  0,1,0,0,  0,0,1,0,  0,0,0,1 };
     float  fx = 1.f, fy = 1.f, tan_fx = 1.f, tan_fy = 1.f;
 
     // 上游对 conic 的梯度（只用到 0,1,3 下标）
     float  h_dLdConic[4] = { 1.f, 1.f, 0.f, 1.f };
 
     /* ---------- 2. 设备端 buffer ---------- */
     float3 *d_means, *d_dLdMeans;
     int    *d_radii;
     float  *d_cov3Ds, *d_view, *d_dLdConic, *d_dLdCov;
     float4 *d_conic_base, *d_conic_eps;
 
     hipMalloc(&d_means, sizeof(h_means));
     hipMalloc(&d_radii, sizeof(h_radii));
     hipMalloc(&d_cov3Ds, sizeof(h_cov3Ds));
     hipMalloc(&d_view, 16*sizeof(float));
     hipMalloc(&d_dLdConic, sizeof(h_dLdConic));
     hipMalloc(&d_dLdMeans, sizeof(float3)*P);
     hipMalloc(&d_dLdCov,   sizeof(float)*6*P);
     hipMalloc(&d_conic_base, sizeof(float4)*P);
     hipMalloc(&d_conic_eps,  sizeof(float4)*P);
 
     hipMemcpy(d_means,  h_means,  sizeof(h_means),  hipMemcpyHostToDevice);
     hipMemcpy(d_radii,  h_radii,  sizeof(h_radii),  hipMemcpyHostToDevice);
     hipMemcpy(d_cov3Ds, h_cov3Ds,sizeof(h_cov3Ds), hipMemcpyHostToDevice);
     hipMemcpy(d_view,   h_view,   16*sizeof(float), hipMemcpyHostToDevice);
     hipMemcpy(d_dLdConic, h_dLdConic,sizeof(h_dLdConic), hipMemcpyHostToDevice);
 
     /* ---------- 3. forward 基线 ---------- */
     ForwardKernel<<<1,1>>>(P, d_means, d_cov3Ds,
                            fx, fy, tan_fx, tan_fy,
                            d_view, d_conic_base);
 
     /* ---------- 4. backward（解析梯度） ---------- */
     computeCov2DCUDA<<<1,32>>>(P,
         d_means, d_radii, d_cov3Ds,
         fx, fy, tan_fx, tan_fy,
         d_view, d_dLdConic,
         d_dLdMeans, d_dLdCov);
 
     hipDeviceSynchronize();
     hipError_t err = hipGetLastError();
     if (err != hipSuccess) {
         printf("CUDA launch error: %s\n", hipGetErrorString(err));
         return -1;
     }
 
     /* 拷回解析梯度 */
     float3 g_mean;  float g_cov[6];
     hipMemcpy(&g_mean, d_dLdMeans, sizeof(g_mean), hipMemcpyDeviceToHost);
     hipMemcpy(g_cov,   d_dLdCov,   sizeof(g_cov), hipMemcpyDeviceToHost);
 
     /* ---------- 5. 有限差分：只验证 μ 与 Σzz ---------- */
     const float eps = 1e-3f;
     auto fd = [&](auto setter)
     {
         setter(+eps);
         hipMemcpy(d_means,  h_means,  sizeof(h_means),  hipMemcpyHostToDevice);
         hipMemcpy(d_cov3Ds, h_cov3Ds,sizeof(h_cov3Ds), hipMemcpyHostToDevice);
         ForwardKernel<<<1,1>>>(P, d_means, d_cov3Ds, fx, fy, tan_fx, tan_fy, d_view, d_conic_eps);
         float4 pos;  hipMemcpy(&pos, d_conic_eps, sizeof(float4), hipMemcpyDeviceToHost);
 
         setter(-eps);
         hipMemcpy(d_means,  h_means,  sizeof(h_means),  hipMemcpyHostToDevice);
         hipMemcpy(d_cov3Ds, h_cov3Ds,sizeof(h_cov3Ds), hipMemcpyHostToDevice);
         ForwardKernel<<<1,1>>>(P, d_means, d_cov3Ds, fx, fy, tan_fx, tan_fy, d_view, d_conic_eps);
         float4 neg;  hipMemcpy(&neg, d_conic_eps, sizeof(float4), hipMemcpyDeviceToHost);
 
         setter(0.f);                                     // 还原
         float Lpos = pos.x*h_dLdConic[0] + pos.y*h_dLdConic[1] + pos.w*h_dLdConic[3];
         float Lneg = neg.x*h_dLdConic[0] + neg.y*h_dLdConic[1] + neg.w*h_dLdConic[3];
         return (Lpos - Lneg)/(2*eps);
     };
 
     float g_fd_mu[3];
     g_fd_mu[0] = fd([&](float d){ h_means[0].x += d; });
     g_fd_mu[1] = fd([&](float d){ h_means[0].y += d; });
     g_fd_mu[2] = fd([&](float d){ h_means[0].z += d; });
     float g_fd_covzz = fd([&](float d){ h_cov3Ds[5] += d; });
 
     /* ---------- 6. 打印并判断 ---------- */
     printf("\nAnalytic   dL/dμ = (%f %f %f)\n", g_mean.x, g_mean.y, g_mean.z);
     printf("FiniteDiff dL/dμ = (%f %f %f)\n", g_fd_mu[0], g_fd_mu[1], g_fd_mu[2]);
     printf("Analytic   dL/dΣzz = %f\n", g_cov[5]);
     printf("FiniteDiff dL/dΣzz = %f\n", g_fd_covzz);
 
     const float tol = 1e-2f;
     bool pass = fabsf(g_mean.x-g_fd_mu[0])<tol &&
                 fabsf(g_mean.y-g_fd_mu[1])<tol &&
                 fabsf(g_mean.z-g_fd_mu[2])<tol &&
                 fabsf(g_cov[5] -g_fd_covzz)<tol;
 
     printf("\nGradient check: %s\n", pass ? "PASS" : "FAIL");
     return pass ? 0 : 1;
 }
 