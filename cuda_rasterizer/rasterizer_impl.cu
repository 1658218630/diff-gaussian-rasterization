#include "hip/hip_runtime.h"
#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <thrust/sequence.h>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,
	uint64_t* gaussian_keys_unsorted,
	uint32_t* gaussian_values_unsorted,
	int* radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t*)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
		if (idx == L - 1)
			ranges[currtile].y = L;
	}
}

CudaRasterizer::RasterizerImpl::RasterizerImpl(int resizeMultiplier)
	: resizeMultiplier(resizeMultiplier)
{}

// Instantiate rasterizer
CudaRasterizer::Rasterizer* CudaRasterizer::Rasterizer::make(int resizeMultiplier)
{
	printf("Version 17\n");
	return new CudaRasterizer::RasterizerImpl(resizeMultiplier);
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::RasterizerImpl::markVisible(
		int P,
		float* means3D,
		float* viewmatrix,
		float* projmatrix,
		bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
void CudaRasterizer::RasterizerImpl::forward(
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	int* radii)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	// Dynamically resize auxiliary buffers during training
	if (P > maxP)
	{
		maxP = resizeMultiplier * P;
		cov3D.resize(maxP * 6);
		rgb.resize(maxP * 3);
		tiles_touched.resize(maxP);
		point_offsets.resize(maxP);
		clamped.resize(3 * maxP);

		depths.resize(maxP);
		means2D.resize(maxP);
		conic_opacity.resize(maxP);

		hipcub::DeviceScan::InclusiveSum(nullptr, scan_size, tiles_touched.data().get(), tiles_touched.data().get(), maxP);
		scanning_space.resize(scan_size);
	}

	if (radii == nullptr)
	{
		internal_radii.resize(maxP);
		radii = internal_radii.data().get();
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	if (width * height > maxPixels)
	{
		maxPixels = width * height;
		accum_alpha.resize(maxPixels);
		n_contrib.resize(maxPixels);
		ranges.resize(tile_grid.x * tile_grid.y);
	}

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		clamped.data().get(),
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D.data().get(),
		depths.data().get(),
		cov3D.data().get(),
		rgb.data().get(),
		conic_opacity.data().get(),
		tile_grid,
		tiles_touched.data().get(),
		prefiltered
		);

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	hipcub::DeviceScan::InclusiveSum(scanning_space.data().get(), scan_size,
		tiles_touched.data().get(), point_offsets.data().get(), P);

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_needed;
	hipMemcpy(&num_needed, point_offsets.data().get() + P - 1, sizeof(int), hipMemcpyDeviceToHost);
	if (num_needed > point_list_keys_unsorted.size())
	{
		point_list_keys_unsorted.resize(2 * num_needed);
		point_list_keys.resize(2 * num_needed);
		point_list_unsorted.resize(2 * num_needed);
		point_list.resize(2 * num_needed);
		hipcub::DeviceRadixSort::SortPairs(
			nullptr, sorting_size,
			point_list_keys_unsorted.data().get(), point_list_keys.data().get(),
			point_list_unsorted.data().get(), point_list.data().get(),
			2 * num_needed);
		list_sorting_space.resize(sorting_size);
	}

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P, 
		means2D.data().get(),
		depths.data().get(), 
		point_offsets.data().get(), 
		point_list_keys_unsorted.data().get(), 
		point_list_unsorted.data().get(), 
		radii,
		tile_grid
		);

	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	hipcub::DeviceRadixSort::SortPairs(
		list_sorting_space.data().get(),
		sorting_size,
		point_list_keys_unsorted.data().get(), point_list_keys.data().get(),
		point_list_unsorted.data().get(), point_list.data().get(),
		num_needed, 0, 32 + bit);

	hipMemset(ranges.data().get(), 0, tile_grid.x * tile_grid.y * sizeof(uint2));

	// Identify start and end of per-tile workloads in sorted list
	identifyTileRanges << <(num_needed + 255) / 256, 256 >> > ( 
		num_needed, 
		point_list_keys.data().get(), 
		ranges.data().get()
		);

	// Let each tile blend its range of Gaussians independently in parallel
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : rgb.data().get();
	FORWARD::render(
		tile_grid, block,
		ranges.data().get(),
		point_list.data().get(),
		width, height,
		means2D.data().get(),
		feature_ptr,
		conic_opacity.data().get(),
		accum_alpha.data().get(),
		n_contrib.data().get(),
		background,
		out_color);
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass
void CudaRasterizer::RasterizerImpl::backward(
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot)
{
	if (radii == nullptr)
	{
		radii = internal_radii.data().get();
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : rgb.data().get();
	BACKWARD::render(
		tile_grid,
		block,
		ranges.data().get(),
		point_list.data().get(),
		width, height,
		background,
		means2D.data().get(),
		conic_opacity.data().get(),
		color_ptr,
		accum_alpha.data().get(),
		n_contrib.data().get(),
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor);

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : cov3D.data().get();
	BACKWARD::preprocess(P, D, M,
		(float3*)means3D, 
		radii,
		shs,
		clamped.data().get(),
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot);
}

CudaRasterizer::RasterizerImpl::~RasterizerImpl()
{
}